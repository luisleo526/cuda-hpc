#include <iostream>

int main() {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Size (MB): %zu\n", prop.totalGlobalMem / 1024 / 1024);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
        printf("  Max Threads Dim: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1],
               prop.maxThreadsDim[2]);
        printf("  Max Grid Size: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1],
               prop.maxGridSize[2]);
        printf(" Max Shared Memory Per Block: %d \n\n", prop.sharedMemPerBlock);

    }
}
