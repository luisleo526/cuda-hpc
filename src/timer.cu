//
// Created by leo on 7/8/23.
//

#include "timer.cuh"


Timer::Timer() {
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
}

Timer::~Timer() {
    hipEventDestroy(event_start);
    hipEventDestroy(event_stop);
}

void Timer::start(hipStream_t stream) {
    hipEventRecord(event_start, stream);
}

float Timer::elapsed_time(hipStream_t stream) {
    hipEventRecord(event_stop, stream);
    hipEventSynchronize(event_stop);
    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, event_start, event_stop);
    return elapsed_time;
}

